
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c)
{
    for(int idx = 0; idx < N; idx++) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void device_add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void fill_array(int *data) {
    for(int idx = 0; idx < N; idx++) {
        data[idx] = idx;
    }
}

void print_output(int *a, int *b, int *c)
{
    for(int idx = 0; idx < N; idx++) {
        printf("%d + %d = %d\n", a[idx], b[idx], c[idx]);
    }
}


int main(void)
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;   // a, b, c をデバイスにコピーするための変数
    int size = N * sizeof(int);

    // ホスト側でメモリを確保
    a = (int *)malloc(size);
    fill_array(a);
    b = (int *)malloc(size);
    fill_array(b);
    c = (int *)malloc(size);

    // a, b, c に対応する GPU メモリを確保
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // ホストからデバイスにコピー
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // 計算の実行
    device_add<<<N, 1>>>(d_a, d_b, d_c);

    // 結果をデバイスからホストにコピー
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    print_output(a, b, c);
    free(a);
    free(b);
    free(c);

    // GPU メモリの解放
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}