
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void init(int n, float *x, float *y) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}
 
// 2つの配列を足し合わせる CUDA カーネル
__global__ void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}
 
int main(void)
{
    int N = 1<<20;
    float *x, *y;

    // Unified memory の割当 (CPU と GPU からアクセス可能)
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // GPU 上で 1M この要素からなるカーネルを立ち上げる
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    init<<<numBlocks, blockSize>>>(N, x, y);
    add<<<numBlocks, blockSize>>>(N, x, y);

    // カーネルの実行が終わるまで待つ
    hipDeviceSynchronize();

    // エラーがないかチェック (全て 3.0f になっているはず)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // メモリの解放
    hipFree(x);
    hipFree(y);

    return 0;
}
