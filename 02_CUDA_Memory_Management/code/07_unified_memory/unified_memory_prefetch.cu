#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
 
// 2つの配列を足し合わせる CUDA カーネル
__global__ void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}
 
int main(void)
{
    int N = 1<<20;
    float *x, *y;
    int device = -1;

    // Unified memory の割当 (CPU と GPU からアクセス可能)
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // ホスト側で配列を初期化
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // GPU が unified memory のメモリをプリフェッチする
    hipGetDevice(&device);
    hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
    hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);

    // GPU 上で 1M この要素からなるカーネルを立ち上げる
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);
    // ホストがメモリをプリフェッチする
    hipMemPrefetchAsync(y, N*sizeof(float), hipCpuDeviceId, NULL);
    // カーネルの実行が終わるまで待つ
    hipDeviceSynchronize();

    // エラーがないかチェック (全て 3.0f になっているはず)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // メモリの解放
    hipFree(x);
    hipFree(y);

    return 0;
}
